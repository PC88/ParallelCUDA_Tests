#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "Kernel.h"
#include <stdio.h>
#include <math.h>
#define TPB 32

__device__ float scale(int i, int n)
{
	return ((float)i) / (n - 1);
}

__device__ float distance(float x1, float x2)
{
	return sqrt((x2 - x1) * (x2 - x1));
}

// converted from serial app
__global__ void distanceKernel(float* d_out, float ref, int len)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const float x = scale(i, len);
	d_out[i] = distance(x, ref);
	printf("i = %2d: dist from %f to %f.\n", i, ref, x, d_out[i]);
}

__global__ void distanceArray(float* out, float* in, float ref, int len)
{
	// Declare pointers to device arrays
	float* d_in = 0;
	float* d_out = 0;

	// Allowcate memory for device arrays
	hipMalloc(&d_in, len * sizeof(float));
	hipMalloc(&d_out, len * sizeof(float));

	// Copy input data from host to device
	hipMemcpy(d_in, in, len * sizeof(float), hipMemcpyHostToDevice);

	// launch kernel to compute and store distance values
	distanceKernel<<<len / TPB, TPB >>>(d_out, d_in, ref);

	// Copy results from device to host
	hipMemcpy(out, d_out, len * sizeof(float), hipMemcpyDeviceToHost);

	// free the memory allowcated for device arrays
	hipFree(d_in);
	hipFree(d_out);
}

