
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>

#define N 64 // constant array lenght
#define TPB 32 // threads per block

__device__ float scale(int i, int n)
{
	return ((float)i) / (n - 1);
}

__device__ float distance(float x1, float x2)
{
	return sqrt((x2 - x1) * (x2 - x1));
}

// converted from serial app
__global__ void distanceKernel(float* d_out, float ref, int len)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const float x = scale(i, len);
	d_out[i] = distance(x, ref);
	printf("i = %2d: dist from %f to %f.\n", i, ref, x, d_out[i]);
}


int main()
{
	const float ref = 0.5f;

	// Declare a pointer for an array of floats
	float* d_out = 0;

	// Allowcate device memory to store the output array
	hipMalloc(&d_out, N * sizeof(float));

	// Launch Kernel to compute and store distance values
	distanceKernel<<<N / TPB, TPB>>>(d_out, ref, N);

	// Free the memory
	hipFree(d_out);

    return 0;
}
